
#include <hip/hip_runtime.h>
#include "stdio.h"

const int N=10000;

__global__ void add_arrays_gpu( float *in1, float *in2, float *out, int Ntot)
{
int idx=blockIdx.x*blockDim.x+threadIdx.x;
if (idx<N)
out[idx]=in1[idx]+in2[idx];
}

int main()
{
/* pointers to host memory */
float *a, *b, *c;
/* pointers to device memory */
float *a_d, *b_d, *c_d;
int i;

/* Allocate arrays a, b and c on host*/
a = (float*) malloc(N*sizeof(float));
b = (float*) malloc(N*sizeof(float));
c = (float*) malloc(N*sizeof(float));

/* Allocate arrays a_d, b_d and c_d on device*/
hipMalloc ((void **) &a_d, sizeof(float)*N);
hipMalloc ((void **) &b_d, sizeof(float)*N);
hipMalloc ((void **) &c_d, sizeof(float)*N);

/* Initialize arrays a and b */
for (i=0; i<N;i++)
{
a[i]= (float) i;
b[i]= -(float) i;
}


/* Copy data from host memory to device memory */
hipMemcpy(a_d, a, sizeof(float)*N, hipMemcpyHostToDevice);
hipMemcpy(b_d, b, sizeof(float)*N, hipMemcpyHostToDevice);

/* Compute the execution configuration */
int block_size=8;
dim3 dimBlock(block_size);
dim3 dimGrid ( ceil((float(N)/(float)dimBlock.x)) );

/* Add arrays a and b, store result in c */
add_arrays_gpu<<< dimGrid, dimBlock >>>(a_d, b_d, c_d, N);

/* Copy data from deveice memory to host memory */
hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

/* Print c */
for (i=0; i<N;i++)
printf(" c[%d]=%f\n",i,c[i]);

/* Free the memory */
free(a); free(b); free(c);
hipFree(a_d); hipFree(b_d);hipFree(c_d);

}
